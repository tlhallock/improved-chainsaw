#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void sumArray(float *in, float *out, int N);

int main() {
    int N = 1024; // Size of the array
    size_t bytes = N * sizeof(float);

    float *h_in = (float*) malloc(bytes);
    float *h_out = (float*) malloc(sizeof(float)); // Output variable on host
    float *d_in, *d_out; // Device input and output pointers

    // Initialize input array with some values
    for(int i = 0; i < N; i++) {
        h_in[i] = 2.0f;
    }

    // Allocate device memory
    hipMalloc((void**)&d_in, bytes);
    hipMalloc((void**)&d_out, sizeof(float));

    // Transfer data from host to device
    hipMemcpy(d_in, h_in, bytes, hipMemcpyHostToDevice);

    // Kernel launch
    int threadsPerBlock = 256;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
    sumArray<<<blocksPerGrid, threadsPerBlock>>>(d_in, d_out, N);

    // Copy result back to host
    hipMemcpy(h_out, d_out, sizeof(float), hipMemcpyDeviceToHost);

    // Print result
    printf("Result: %f\n", *h_out);

    // Free device memory
    hipFree(d_in);
    hipFree(d_out);

    // Free host memory
    free(h_in);
    free(h_out);

    return 0;
}

__global__ void sumArray(float *in, float *out, int N) {
    // Your implementation here
}
